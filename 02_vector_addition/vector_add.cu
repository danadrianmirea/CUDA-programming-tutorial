/*
STEPS 
1. Allocate host memory and initialized host data e.g. malloc
2. Allocate device memory e.g cudaMalloc
3. Transfer input data from host to device memory e.g cudaMemcpy
4. Execute kernels
5. Transfer output from device memory to host
6. Free cuda memory e.g. cudaFree
*/



#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 
    float *d_a, *d_b, *d_out;

    // 1. Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // 1. Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // 2. Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // 3. Transfer input data from host to device
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // 4. Kernel launch
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);      //use only one thread

    // 5. Transfer output from device memory to host
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // 6. Free cuda memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
}

